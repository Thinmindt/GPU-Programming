#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void preScanKernel(float *inout, unsigned size, float *sum)
{
	__shared__ float in_s[BLOCK_SIZE*2];
	int i, idx, offset;

	idx = 2*blockIdx.x*BLOCK_SIZE+threadIdx.x;
	
		int blid = blockIdx.x * BLOCK_SIZE * 2;
		int thid = threadIdx.x;

        offset=1;
	if(idx < size)
		in_s[threadIdx.x] = inout[idx];
	else
		in_s[threadIdx.x] = 0.0f;

	if(idx+BLOCK_SIZE < size)
		in_s[threadIdx.x+BLOCK_SIZE] = inout[idx+BLOCK_SIZE];
	else
		in_s[threadIdx.x+BLOCK_SIZE] = 0.0f;


	/*for (int d = (2 * BLOCK_SIZE)>>1; d > 0; d >>=1) {
		__syncthreads();

		if (thid < d) {
			int ai = offset * (2*thid+1) -1;
			int bi = offset * (2*thid+2) -1;

			in_s[bi] += in_s[ai];
		}

		offset *= 2;
	}*/


	for(i=BLOCK_SIZE, offset=1; i>0; i>>=1, offset<<=1) {
		__syncthreads();

		if(threadIdx.x < i)
			in_s[offset*(2*threadIdx.x+2)-1] +=
				in_s[offset*(2*threadIdx.x+1)-1];
	}

	if (thid == 0) {
		if (sum != NULL)
			sum[blockIdx.x] = in_s[BLOCK_SIZE * 2 - 1];

		in_s[BLOCK_SIZE * 2 - 1] = 0;
	}

/*	for (int d = 1; d < BLOCK_SIZE * 2; d*= 2) {
		offset >>= 1;
		__syncthreads();

		if (thid < d) {
			int ai = offset * (2 * thid+1) -1;
			int bi = offset * (2 * thid+2) -1;

			float t = in_s[ai];
			in_s[ai] = in_s[bi];
			in_s[bi] += t;
		}
	}
*/


	for(i=1, offset=BLOCK_SIZE ; i<=BLOCK_SIZE; i<<=1, offset>>=1) {
		__syncthreads();

		if(threadIdx.x < i) {
			float t = in_s[offset*(2*threadIdx.x+1)-1];
			in_s[offset*(2*threadIdx.x+1)-1] =
				in_s[offset*(2*threadIdx.x+2)-1];
			in_s[offset*(2*threadIdx.x+2)-1] += t;
		}
	}

	__syncthreads();

		if (thid + blid < size)
			inout[blid + thid] = in_s[thid];
		if (thid + BLOCK_SIZE + blid < size)
			inout[thid+BLOCK_SIZE + blid] = in_s[thid+BLOCK_SIZE];
}


__global__ void addKernel(float *inout, float *sum, unsigned size)
{
	int thid = threadIdx.x;
	int blid = 2 * blockIdx.x * BLOCK_SIZE;
	
	if(blid + thid < size) 
		inout[blid + thid] += sum[blockIdx.x];
	
	if(blid + BLOCK_SIZE + thid < size)
		inout[blid + BLOCK_SIZE + thid] += sum[blockIdx.x];
}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void preScan(float *inout, unsigned in_size)
{
	float *sum;
	unsigned num_blocks;
	hipError_t cuda_ret;
	dim3 dim_grid, dim_block;

	num_blocks = in_size/(BLOCK_SIZE*2);
	if(in_size%(BLOCK_SIZE*2) !=0) num_blocks++;

	dim_block.x = BLOCK_SIZE; dim_block.y = 1; dim_block.z = 1;
	dim_grid.x = num_blocks; dim_grid.y = 1; dim_grid.z = 1;

	if(num_blocks > 1) {
		cuda_ret = hipMalloc((void**)&sum, num_blocks*sizeof(float));
		if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

		preScanKernel<<<dim_grid, dim_block>>>(inout, in_size, sum);
		preScan(sum, num_blocks);
		addKernel<<<dim_grid, dim_block>>>(inout, sum, in_size);

		hipFree(sum);
	}
	else
		preScanKernel<<<dim_grid, dim_block>>>(inout, in_size, NULL);
}

