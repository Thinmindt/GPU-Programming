/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    float CVal = 0;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    for (int i = 0; i < (TILE_SIZE + k - 1)/TILE_SIZE; i++) {
        if (i * TILE_SIZE + threadIdx.x < k && row < m)
            sharedA[threadIdx.y][threadIdx.x] = A[row*k + i*TILE_SIZE + threadIdx.x];
        else
            sharedA[threadIdx.y][threadIdx.x] = 0.0;

        if (i * TILE_SIZE + threadIdx.y < k && col < n)
            sharedB[threadIdx.y][threadIdx.x] = B[(i*TILE_SIZE + threadIdx.y)*n+col];
        else
            sharedB[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int j = 0; j < TILE_SIZE; ++j)
            CVal += sharedA[threadIdx.y][j] * sharedB[j][threadIdx.x];

        __syncthreads();
    }


    if (row < m && col < n) {
        C[row * n + col] = CVal;

    }






























}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((int)ceil((float)n/BLOCK_SIZE),(int)ceil((float)m/BLOCK_SIZE));


    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE

    mysgemm<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, A, B, C);


}


