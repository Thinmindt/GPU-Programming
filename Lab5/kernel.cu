
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE

    __shared__ float sdata[512];
    int i = blockIdx.x * 2 * blockDim.x + threadIdx.x; 
    int tx = threadIdx.x;

    float x = 0;
    float y = 0;
    if (i < size) {
    	x = in[i];
    }

    int secondLoad = i + blockDim.x;
    if (secondLoad < size) {
        y = in[secondLoad];
    }
    sdata[tx] = x + y;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    	if (tx < offset)
    		sdata[tx] += sdata[tx + offset];
    	__syncthreads();
    }

    if (tx == 0) {
    	out[blockIdx.x] = sdata[0];
    }
}
